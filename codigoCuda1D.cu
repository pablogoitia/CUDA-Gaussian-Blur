#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image.h"
#include "lib/stb_image_write.h"

#include <stdio.h>

float *createFilter(int width)
{
    const float sigma = 2.f; // Standard deviation of the Gaussian distribution.

    const int half = width / 2;
    float sum = 0.f;

    // Create convolution matrix
    float *res = (float *)malloc(width * width * sizeof(float));

    // Calculate filter sum first
    for (int r = -half; r <= half; ++r)
    {
        for (int c = -half; c <= half; ++c)
        {
            // e (natural logarithm base) to the power x, where x is what's in the brackets
            float weight = expf(-static_cast<float>(c * c + r * r) / (2.f * sigma * sigma));
            int idx = (r + half) * width + c + half;

            res[idx] = weight;
            sum += weight;
        }
    }

    // Normalize weight: sum of weights must equal 1
    float normal = 1.f / sum;

    for (int r = -half; r <= half; ++r)
    {
        for (int c = -half; c <= half; ++c)
        {
            int idx = (r + half) * width + c + half;

            res[idx] *= normal;
        }
    }
    return res;
}

// Copmute gaussian blur per channel on the GPU.
// Call this function for each of red, green, and blue channels
// Returns blurred channel.
__global__ void ComputeConvolution(unsigned char *const blurredChannel, const unsigned char *const inputChannel, int rows, int cols, float *filter, int filterWidth)
{
    // Filter width should be odd as we are calculating average blur for a pixel plus some offset in all directions

    const int half = filterWidth / 2;
    const int width = cols - 1;
    const int height = rows - 1;

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Compute blur(int c = 0; c < cols; ++c)
    float blur = 0.f;

    // Average pixel color summing up adjacent pixels.
    if (tid < rows * cols)
    {
        for (int i = -half; i <= half; ++i)
        {
            for (int j = -half; j <= half; ++j)
            {
                // Clamp filter to the image border
                int h = min(max(r + i, 0), height);
                int w = min(max(c + j, 0), width);

                // Blur is a product of current pixel value and weight of that pixel.
                // Remember that sum of all weights equals to 1, so we are averaging sum of all pixels by their weight.
                int idx = w + cols * h; // current pixel index
                float pixel = static_cast<float>(inputChannel[idx]);

                idx = (i + half) * filterWidth + j + half;
                float weight = filter[idx];

                blur += pixel * weight;
            }
        }

        blurredChannel[c + cols * r] = static_cast<unsigned char>(blur);
    }
}

void GaussianBlur(uchar4 *const modifiedImage, const uchar4 *const rgba, int rows, int cols, float *filter, int filterWidth, dim3 block, dim3 grid)
{
    const int numPixels = rows * cols;
    const int channelSize = numPixels * sizeof(unsigned char);

    // Create channel variables
    unsigned char *red = new unsigned char[numPixels];
    unsigned char *green = new unsigned char[numPixels];
    unsigned char *blue = new unsigned char[numPixels];

    unsigned char *redBlurred = new unsigned char[numPixels];
    unsigned char *greenBlurred = new unsigned char[numPixels];
    unsigned char *blueBlurred = new unsigned char[numPixels];

    // GPU Device Memory
    unsigned char *channelGPU, *blurredChannelGPU;
    float *filterGPU;
    hipMalloc(&filterGPU, filterWidth * filterWidth * sizeof(float));
    hipMalloc(&channelGPU, channelSize);
    hipMalloc(&blurredChannelGPU, channelSize);

    // Separate RGBAimage into red, green, and blue components
    for (int p = 0; p < numPixels; ++p)
    {
        uchar4 pixel = rgba[p];

        red[p] = pixel.x;
        green[p] = pixel.y;
        blue[p] = pixel.z;
    }

    // Compute convolution for each individual channel
    hipMemcpy((void *)filterGPU, (void *)filter, filterWidth * filterWidth * sizeof(float), hipMemcpyHostToDevice);

    /* Red channel */
    hipMemcpy((void *)channelGPU, (void *)red, channelSize, hipMemcpyHostToDevice);
    ComputeConvolution<<<grid, block>>>(blurredChannelGPU, channelGPU, rows, cols, filterGPU, filterWidth);
    hipDeviceSynchronize();

    hipMemcpy((void *)redBlurred, (void *)blurredChannelGPU, channelSize, hipMemcpyDeviceToHost);

    /* Green channel */
    hipMemcpy((void *)channelGPU, (void *)green, channelSize, hipMemcpyHostToDevice);
    ComputeConvolution<<<grid, block>>>(blurredChannelGPU, channelGPU, rows, cols, filterGPU, filterWidth);
    hipDeviceSynchronize();

    hipMemcpy((void *)greenBlurred, (void *)blurredChannelGPU, channelSize, hipMemcpyDeviceToHost);

    /* Blue channel */
    hipMemcpy((void *)channelGPU, (void *)blue, channelSize, hipMemcpyHostToDevice);
    ComputeConvolution<<<grid, block>>>(blurredChannelGPU, channelGPU, rows, cols, filterGPU, filterWidth);
    hipDeviceSynchronize();

    hipMemcpy((void *)blueBlurred, (void *)blurredChannelGPU, channelSize, hipMemcpyDeviceToHost);

    // Recombine channels back into an RGBAimage setting alpha to 255, or fully opaque
    for (int p = 0; p < numPixels; ++p)
    {
        unsigned char r = redBlurred[p];
        unsigned char g = greenBlurred[p];
        unsigned char b = blueBlurred[p];

        modifiedImage[p] = make_uchar4(r, g, b, 255);
    }

    hipFree(filterGPU);
    hipFree(channelGPU);
    hipFree(blurredChannelGPU);

    delete[] red;
    delete[] green;
    delete[] blue;
    delete[] redBlurred;
    delete[] greenBlurred;
    delete[] blueBlurred;
}

// Main entry into the application
int main(int argc, char **argv)
{
    char *imagePath;
    char *outputPath;

    int height, width, bpp, channels = 4;
    uchar4 *originalImage, *blurredImage;

    int filterWidth = 9;
    float *filter = createFilter(filterWidth);

    struct timespec start, end;

    // GPU variables
    dim3 block, grid;

    if (argc > 2)
    {
        imagePath = argv[1];
        outputPath = argv[2];
    }
    else
    {
        printf("Please provide input and output image files as arguments to this application.");
        exit(1);
    }

    // Read the image
    uint8_t *rgb_image = stbi_load(imagePath, &width, &height, &bpp, channels);

    if (rgb_image == NULL)
        printf("Could not load image file: %s\n", imagePath);

    // Allocate and copy
    originalImage = (uchar4 *)malloc(width * height * sizeof(uchar4));
    blurredImage = (uchar4 *)malloc(width * height * sizeof(uchar4));
    printf("Width:%d, Height:%d Size(in Bytes):%d\n", width, height, width * height * bpp * channels);
    for (int i = 0; i < width * height * channels; i++)
    {
        int mod = i % channels;
        switch (mod)
        {
        case 0:
            originalImage[i / channels].x = rgb_image[i];
            break;
        case 1:
            originalImage[i / channels].y = rgb_image[i];
            break;
        case 2:
            originalImage[i / channels].z = rgb_image[i];
            break;
        case 3:
            originalImage[i / channels].w = rgb_image[i];
            break;
        }
    }

    // Calculate block and grid dimensions based on the parameters and the image dimensions
    if (argc > 3)
        block.x = atoi(argv[3]);
    else
        block.x = 128;

    grid.x = ceil((double)width * height / block.x);
    printf("block=%d, grid=%d\n", block.x, grid.x);

    // Apply the gaussian blur over the image with the given filter
    clock_gettime(CLOCK_MONOTONIC, &start);
    GaussianBlur(blurredImage, originalImage, height, width, filter, filterWidth, block, grid);
    clock_gettime(CLOCK_MONOTONIC, &end);

    // Write the image back to disk
    stbi_write_jpg(outputPath, width, height, 4, blurredImage, 100);

    // Debug info
    printf("Time: %Lf\n", (long double)((end.tv_sec - start.tv_sec) * 1000000000 + (end.tv_nsec - start.tv_nsec)) / 1000000000);
    printf("Done!\n");
    return 0;
}
